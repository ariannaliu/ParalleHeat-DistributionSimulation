#include "hip/hip_runtime.h"
#include <graphic/graphic.hpp>
#include <imgui_impl_sdl.h>
#include <cstring>
#include <chrono>
#include <hdist/hdist.hpp>
#include <iostream>
#include <random>
#include <utility>
#include <time.h>
#include <cstdlib>
#include <cmath>
#include <sys/time.h>


using namespace std;

template<typename ...Args>
void UNUSED(Args &&... args [[maybe_unused]]) {}



__device__
int getBlockId() {
  return blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
}

__device__
int getLocalThreadId() {
  return (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
}

__device__
int getThreadId() {
  int blockId = getBlockId();
  int localThreadId = getLocalThreadId();
  return blockId * (blockDim.x * blockDim.y * blockDim.z) + localThreadId;
}




// __shared__ bool local_stable = true;
// __device__ bool *local_stable;

__host__ __device__
int get_index_in_array(size_t i, size_t j, int length){
    return i * length + j;
}


__device__
int get_buffer(int current_buffer){
    if (current_buffer == 0){
        return 0;
    }
    return 1;
}

__device__
int get_buffer_alt(int current_buffer){
    if (current_buffer == 0){
        return 1;
    }
    return 0;
}

__device__
int switch_buffer(int current_buffer){
    if (current_buffer == 0){
        return 1;
    }
    return 0;
}

__device__
void update_single(int i, int j, 
                   int room_size, float block_size,
                int source_x, int source_y,
                float source_temp, float border_temp,
                float tolerance, float sor_constant,
                int algo,
                double *data0, double *data1, int length, int current_buffer,
                bool & stable, double & temp){
    if (i == 0 || j == 0 || i == room_size - 1 || j == room_size - 1) {
        temp = border_temp;
    } else if (i == source_x && j == source_y) {
        temp = source_temp;
    } else {
        double sum;
        if(current_buffer == 0){
            sum = data0[get_index_in_array(i + 1, j, length)] + data0[get_index_in_array(i - 1, j, length)]
                        + data0[get_index_in_array(i, j+1, length)] + data0[get_index_in_array(i, j-1, length)];
        }else{
            sum = data1[get_index_in_array(i + 1, j, length)] + data1[get_index_in_array(i - 1, j, length)]
                        + data1[get_index_in_array(i, j+1, length)] + data1[get_index_in_array(i, j-1, length)];
        }
        if(algo == 0){
            temp =  0.25 * sum;
        }else{
            if(current_buffer == 0){
                temp = data0[get_index_in_array(i, j, length)] + (1.0 / sor_constant)*(sum - 4.0 * data0[get_index_in_array(i, j, length)]);
            }else{
                temp = data1[get_index_in_array(i, j, length)] + (1.0 / sor_constant)*(sum - 4.0 * data1[get_index_in_array(i, j, length)]);
            }
        }
    }
    if(current_buffer == 0){
        stable = std::fabs(data0[get_index_in_array(i, j, length)] - temp) < tolerance;   
    }else{
        stable = std::fabs(data1[get_index_in_array(i, j, length)] - temp) < tolerance;   
    }
}

__global__
void calculate(int room_size, float block_size,
                int source_x, int source_y,
                float source_temp, float border_temp,
                float tolerance, float sor_constant,
                int algo,
                double *data0, double *data1, int length, int current_buffer, int local_size, bool *devicestable){
    bool stabilized = true;
    *devicestable = true;
    
    int thread_id = getThreadId();
    printf("Thread_id is %d \n", thread_id);
    if(algo == 0){
        for (size_t i = (thread_id)*local_size; i < (thread_id+1)*local_size; ++i) {
            for (size_t j = 0; j < room_size; ++j) {
                if(get_index_in_array(i, j, length)<room_size*room_size){

                    bool t_stable;
                    double t_temp;
                    update_single(i,j,room_size, block_size, 
                                source_x, source_y, source_temp, border_temp,
                                tolerance, sor_constant, algo, data0, data1, length, current_buffer,
                                t_stable, t_temp);
                    stabilized &= t_stable;
                    if(current_buffer == 0){
                        data1[get_index_in_array(i, j, length)] = t_temp;
                    }else{
                        data0[get_index_in_array(i, j, length)] = t_temp;
                    }
                }
            }
        }
        current_buffer = switch_buffer(current_buffer);
    }else{
        for (auto k : {0, 1}) {
            for (size_t i = (thread_id)*local_size; i < min(room_size, (thread_id+1)*local_size); ++i) {
                for (size_t j = 0; j < room_size; j++) {
                    if(get_index_in_array(i, j, length)<room_size*room_size){
                        bool t_stable;
                        double t_temp;
                        if (k == ((i + j) & 1)) {
                            update_single(i,j,room_size, block_size, 
                                                        source_x, source_y, source_temp, border_temp,
                                                        tolerance, sor_constant, algo, data0, data1, length, current_buffer,
                                                        t_stable, t_temp);
                            stabilized &= t_stable;

                            if(current_buffer == 0){
                                data1[get_index_in_array(i, j, length)] = t_temp;
                            }else{
                                data0[get_index_in_array(i, j, length)] = t_temp;
                            }
                        } else {
                            // grid[{alt, i, j}] = grid[{i, j}];
                            if(current_buffer == 0){
                                data1[get_index_in_array(i, j, length)] = data0[get_index_in_array(i, j, length)];
                            }else{
                                data0[get_index_in_array(i, j, length)] = data1[get_index_in_array(i, j, length)];
                            }
                        }
                    }
                }
            }
            current_buffer = switch_buffer(current_buffer);
        }
    }
    *devicestable &= stabilized;
}





















ImColor temp_to_color(double temp) {
    auto value = static_cast<uint8_t>(temp / 100.0 * 255.0);
    return {value, 0, 255 - value};
}

void init_Grid(size_t size, double border_temp,double source_temp,
                size_t x, size_t y, double *data0, double *data1, 
                int length){
    for (size_t i = 0; i < length; ++i){
        for (size_t j = 0; j < length; ++j){
            int index = get_index_in_array(i, j, length);
            if (i == 0 || j == 0 || i == length - 1 || j == length - 1) {
                data0[index] = border_temp;
            }else if (i == x && j == y){
                data0[index] = source_temp;
            }else{
                data0[index] = 0;
            }
            
        }
    }
}

int main(int argc, char **argv) {
    // UNUSED(argc, argv);
    int thread_num;
    if (argc < 2){
        // if user did not provide the size of array
        // the defualt value is set to be 100
        thread_num = 4;
    }else{
        thread_num = atoi(argv[1]);
    }
    bool first = true;
    bool finished = false;
    // static hdist::State current_state, last_state;
    int room_size_C = 300, room_size_S = 300;
    float block_size_C = 2, block_size_S = 2;
    int source_x_C = room_size_C / 2, source_x_S = room_size_S / 2;
    int source_y_C = room_size_C / 2, source_y_S = room_size_S / 2;
    float source_temp_C = 100, source_temp_S = 100;
    float border_temp_C = 36, border_temp_S = 36;
    float tolerance_C = 0.02, tolerance_S = 0.02;
    float sor_constant_C = 4.0, sor_constant_S = 4.0;
    int algo_C = 0, algo_S = 0;


    static std::chrono::high_resolution_clock::time_point begin, end;
    static const char* algo_list[2] = { "jacobi", "sor" };
    graphic::GraphicContext context{"Assignment 4"};

    double * Hostdata0 = new double[room_size_C*room_size_C];
    double * Hostdata1 = new double[room_size_C*room_size_C];
    bool *Hostdevicestable = new bool;

    double *data0;
    hipMalloc(&data0, sizeof(double) * room_size_C*room_size_C);
    double *data1;
    hipMalloc(&data1, sizeof(double) * room_size_C*room_size_C);
    bool *devicestable;
    hipMalloc(&devicestable, sizeof(bool));

    int length = room_size_C;
    int current_buffer = 0;
    init_Grid(
            static_cast<size_t>(room_size_C),
            border_temp_C, source_temp_C,
            static_cast<size_t>(source_x_C), static_cast<size_t>(source_y_C),
            Hostdata0, Hostdata1, length);
    
    hipMemcpy(data0, Hostdata0, sizeof(double)*room_size_C*room_size_C, hipMemcpyHostToDevice);
    hipMemcpy(data1, Hostdata1, sizeof(double)*room_size_C*room_size_C, hipMemcpyHostToDevice);
    
    context.run([&](graphic::GraphicContext *context [[maybe_unused]], SDL_Window *) {
        auto io = ImGui::GetIO();
        ImGui::SetNextWindowPos(ImVec2(0.0f, 0.0f));
        ImGui::SetNextWindowSize(io.DisplaySize);
        ImGui::Begin("Assignment 4", nullptr,
                     ImGuiWindowFlags_NoMove
                     | ImGuiWindowFlags_NoCollapse
                     | ImGuiWindowFlags_NoTitleBar
                     | ImGuiWindowFlags_NoResize);
        ImDrawList *draw_list = ImGui::GetWindowDrawList();
        ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / ImGui::GetIO().Framerate,
                    ImGui::GetIO().Framerate);
        ImGui::DragInt("Room Size", &room_size_C, 10, 200, 1600, "%d");
        ImGui::DragFloat("Block Size", &block_size_C, 0.01, 0.1, 10, "%f");
        ImGui::DragFloat("Source Temp", &source_temp_C, 0.1, 0, 100, "%f");
        ImGui::DragFloat("Border Temp", &border_temp_C, 0.1, 0, 100, "%f");
        ImGui::DragInt("Source X", &source_x_C, 1, 1, room_size_C - 2, "%d");
        ImGui::DragInt("Source Y", &source_y_C, 1, 1, room_size_C - 2, "%d");
        ImGui::DragFloat("Tolerance", &tolerance_C, 0.01, 0.01, 1, "%f");
        ImGui::ListBox("Algorithm", reinterpret_cast<int *>(&algo_C), algo_list, 2);

        if (algo_C == 1) {
            ImGui::DragFloat("Sor Constant", &sor_constant_C, 0.01, 0.0, 20.0, "%f");
        }

        if (room_size_C != room_size_S) {
            delete[] Hostdata0;
            delete[] Hostdata1;
            Hostdata0 = new double[room_size_C*room_size_C];
            Hostdata1 = new double[room_size_C*room_size_C];
            first = true;
            length = room_size_C;
            current_buffer = 0;
            init_Grid(
                    static_cast<size_t>(room_size_C),
                    border_temp_C,
                    source_temp_C,
                    static_cast<size_t>(source_x_C),
                    static_cast<size_t>(source_y_C), Hostdata0, Hostdata1, length);
            hipMemcpy(data0, Hostdata0, sizeof(double)*(length*length), hipMemcpyHostToDevice);
            hipMemcpy(data1, Hostdata1, sizeof(double)*(length*length), hipMemcpyHostToDevice);
        }

        // if (current_state != last_state) {
        //     last_state = current_state;
        //     finished = false;
        // }
        if (room_size_C != room_size_S){
            room_size_S = room_size_C;
            finished = false;
        }
        if (block_size_C != block_size_S){
            block_size_S = block_size_C;
            finished = false;
        }
        if (source_x_C != source_x_S){
            source_x_S = source_x_C;
            finished = false;
        }
        if (source_y_C != source_y_S){
            source_y_S = source_y_C;
            finished = false;
        }
        if (source_temp_C != source_temp_S){
            source_temp_S = source_temp_C;
            finished = false;
        }
        if (border_temp_C != border_temp_S){
            border_temp_S = border_temp_C;
            finished = false;
        }
        if (tolerance_C != tolerance_S){
            tolerance_S = tolerance_C;
            finished = false;
        }
        if (sor_constant_C != sor_constant_S){
            sor_constant_S = sor_constant_C;
            finished = false;
        }
        if(algo_C != algo_S){
            algo_S = algo_C;
            finished = false;
        }

        if (first) {
            first = false;
            finished = false;
            begin = std::chrono::high_resolution_clock::now();
        }

        int local_size = ceil(double(room_size_C)/thread_num);
        bool Hoststable = true;
        if (!finished) {
            // std::cout<<"Length of data1 "<<sizeof(data1)<<std::endl;
            // std::cout<<"Here1"<<std::endl;
            
            calculate<<<thread_num,1>>>(room_size_C, block_size_C,
                    source_x_C, source_y_C,
                    source_temp_C, border_temp_C,
                    tolerance_C, sor_constant_C,
                    algo_C,
                    data0, data1, length, current_buffer, local_size, devicestable);
            hipDeviceSynchronize();
            hipMemcpy(Hostdata0, data0, sizeof(double)*room_size_C*room_size_C, hipMemcpyDeviceToHost);
            hipMemcpy(Hostdata1, data1, sizeof(double)*room_size_C*room_size_C, hipMemcpyDeviceToHost);
            hipMemcpy(Hostdevicestable, devicestable, sizeof(bool), hipMemcpyDeviceToHost);
            
            finished = *Hostdevicestable;
            if (finished) end = std::chrono::high_resolution_clock::now();
        } else {
            ImGui::Text("stabilized in %ld ns", std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count());
        }

        const ImVec2 p = ImGui::GetCursorScreenPos();
        float x = p.x + block_size_C, y = p.y + block_size_C;
        // std::cout<<"Here5"<<std::endl;
        for (size_t i = 0; i < room_size_C; ++i) {
            for (size_t j = 0; j < room_size_C; ++j) {
                double temp;
                if(current_buffer == 0){
                    temp = Hostdata0[get_index_in_array(i, j, length)];
                }else{
                    temp = Hostdata1[get_index_in_array(i, j, length)];
                }
                auto color = temp_to_color(temp);
                draw_list->AddRectFilled(ImVec2(x, y), ImVec2(x + block_size_C, y + block_size_C), color);
                y += block_size_C;
            }
            x += block_size_C;
            y = p.y + block_size_C;
        }
        ImGui::End();
    });
}
